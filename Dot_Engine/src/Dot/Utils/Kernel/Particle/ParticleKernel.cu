
#include <hip/hip_runtime.h>

#ifdef D_CUDA

#include "ParticleKernel.cuh"
#include "Dot/Core.h"
#include <GL/glew.h>
#include <cuda_gl_interop.h>
#include <iostream>




namespace Dot {

	__global__ void _Init(float3* pos, float3* vel, int count,curandState * state)
	{
		int id = blockDim.x * blockIdx.x + threadIdx.x;
		

		if (id < count)
		{	
			pos[id] = make_float3(0, 0, 0);

			curand_init(1234, id, 0, &state[id]);
			float rand = curand_uniform(&(state[id]));
			vel[id].x = rand/5;

		
			float rand1 = curand_uniform(&(state[id]));
			vel[id].y = -rand1/5;

		
			float rand2 = curand_uniform(&(state[id]));
			vel[id].z = rand2/5;
		}
	}
	__global__ void _UpdateKernel(float3* pos, float3* vel, int count,float dt)
	{
		int id = blockDim.x * blockIdx.x + threadIdx.x;
		
		if (id < count)
		{
			pos[id].x += vel[id].x * dt;
			pos[id].y += vel[id].y * dt;
			pos[id].z += vel[id].z * dt;

			if (abs(pos[id].x) >= 30 || abs(pos[id].y) >= 30 || abs(pos[id].z) >= 30)
			{
				pos[id].x = 0;
				pos[id].y = 0;
				pos[id].z = 0;
			}
		}
	}

	DefaultParticleEffect::DefaultParticleEffect(unsigned int count)
		: m_count(count), m_positions(NULL), m_velocities(NULL)
	{
		CudaErrChk(cudaMallocManaged(&m_positions, sizeof(float3) * m_count));
		CudaErrChk(cudaMallocManaged(&m_velocities, sizeof(float3) * m_count));

		curandState* d_state;
		CudaErrChk(cudaMalloc(&d_state, sizeof(curandState)));

		_Init << <m_count / 256 + 1,256 >> > (m_positions, m_velocities, m_count,d_state);
		
		CudaErrChk(cudaDeviceSynchronize());
		CudaErrChk(cudaFree(d_state));
		
	}

	DefaultParticleEffect::~DefaultParticleEffect()
	{
		CudaErrChk(cudaDeviceSynchronize());
		CudaErrChk(cudaFree(m_positions));
		CudaErrChk(cudaFree(m_velocities));

		cudaDeviceReset();

	}

	void  DefaultParticleEffect::Launch(float dt)
	{			
		_UpdateKernel << <m_count / 256 + 1, 256 >> > (m_positions, m_velocities, m_count, dt);	
		CudaErrChk(cudaDeviceSynchronize());
	}

	__global__ void _TestInit(float3* pos,float3* vel, int count, curandState* state)
	{
		int id = blockDim.x * blockIdx.x + threadIdx.x;

		if (id < count)
		{
			curand_init(1234, id, 0, &state[id]);
			pos[id] = make_float3(0,0,0);
			
			float rand = curand_uniform(&(state[id]));
			vel[id].x = rand / 5;

		
			float rand1 = curand_uniform(&(state[id]));
			vel[id].y = -rand1 / 5;

	
			float rand2 = curand_uniform(&(state[id]));
			vel[id].z = rand2 / 5;
		}
	}

	__global__ void _TestUpdateKernel(float3* pos, float3* vel, int count, float dt)
	{
		int id = blockDim.x * blockIdx.x + threadIdx.x;

		if (id < count)
		{
			pos[id].x += vel[id].x * dt;
			pos[id].y += vel[id].y * dt;
			pos[id].z += vel[id].z * dt;
		}
	}

	TestParticleEffect::TestParticleEffect(unsigned int count)
	: m_count(count)
	{
		glGenVertexArrays(1, &m_VAO);
		glBindVertexArray(m_VAO);
		glGenBuffers(1, &m_VBO);
		glBindBuffer(GL_ARRAY_BUFFER, m_VBO);
		glBufferData(GL_ARRAY_BUFFER, count * sizeof(float3), NULL, GL_DYNAMIC_DRAW);
		
		glEnableVertexAttribArray(0);
		glVertexAttribPointer(0, 3, GL_FLOAT, GL_FALSE, 0, 0);
		glVertexAttribDivisor(0, 1);
		glBindBuffer(GL_ARRAY_BUFFER, 0);
		
		

		CudaErrChk(cudaGLRegisterBufferObject(m_VBO));

		
		curandState* d_state;

		cudaMalloc(&m_velocities, m_count * sizeof(float3));
		CudaErrChk(cudaMalloc(&d_state, sizeof(curandState)));
		CudaErrChk(cudaMalloc(&m_positions, m_count * sizeof(float3)));

		CudaErrChk(cudaGLMapBufferObject((void**)& m_positions, m_VBO));

		_TestInit << <m_count / 256 + 1, 256 >> > (m_positions,m_velocities, m_count,d_state);

		CudaErrChk(cudaGLUnmapBufferObject(m_VBO));

		cudaFree(d_state);

		glBindVertexArray(0);
		cudaDeviceSynchronize();
	}

	TestParticleEffect::~TestParticleEffect()
	{
		CudaErrChk(cudaDeviceSynchronize());
		CudaErrChk(cudaGLUnregisterBufferObject(m_VBO));

		CudaErrChk(cudaFree(m_positions));
		CudaErrChk(cudaFree(m_velocities));

		glDeleteBuffers(1, &m_VBO);
		
		glDeleteBuffers(1, &m_VAO);
	}

	void TestParticleEffect::Render(float dt)
	{

		cudaDeviceSynchronize();
	
		CudaErrChk(cudaGLMapBufferObject((void**)& m_positions, m_VBO));
		
		_TestUpdateKernel << <m_count/256+1, 256 >> > (m_positions,m_velocities, m_count, dt);
	
		CudaErrChk(cudaGLUnmapBufferObject(m_VBO));

		cudaDeviceSynchronize();

		
		glBindVertexArray(m_VAO);

		glDrawArraysInstanced(GL_POINTS, 0,1,m_count);

		glBindVertexArray(0);

	}

	void TestParticleEffect::Launch(float dt)
	{
	}

}


#endif